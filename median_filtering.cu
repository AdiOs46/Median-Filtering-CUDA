#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define MASK_SIZE 3 // Size of the mask or kernel for median filtering

__device__ void bubbleSort(unsigned char *window, int size)
{
    for (int i = 0; i < size - 1; ++i)
    {
        for (int j = 0; j < size - i - 1; ++j)
        {
            if (window[j] > window[j + 1])
            {
                unsigned char temp = window[j];
                window[j] = window[j + 1];
                window[j + 1] = temp;
            }
        }
    }
}

__global__ void medianFilterKernel(const unsigned char *inputImage, unsigned char *outputImage, int width, int height, int channels)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height)
    {
        int index = (row * width + col) * channels;

        // Check boundaries to avoid out-of-bounds access
        if (row >= MASK_SIZE / 2 && row < height - MASK_SIZE / 2 && col >= MASK_SIZE / 2 && col < width - MASK_SIZE / 2)
        {
            unsigned char windowR[MASK_SIZE * MASK_SIZE];
            unsigned char windowG[MASK_SIZE * MASK_SIZE];
            unsigned char windowB[MASK_SIZE * MASK_SIZE];
            int k = 0;

            // Fill the window with pixel values from the neighborhood
            for (int i = -MASK_SIZE / 2; i <= MASK_SIZE / 2; ++i)
            {
                for (int j = -MASK_SIZE / 2; j <= MASK_SIZE / 2; ++j)
                {
                    windowR[k] = inputImage[((row + i) * width + (col + j)) * channels];
                    windowG[k] = inputImage[((row + i) * width + (col + j)) * channels + 1];
                    windowB[k] = inputImage[((row + i) * width + (col + j)) * channels + 2];
                    k++;
                }
            }

            // Sort the window to find the median
            bubbleSort(windowR, MASK_SIZE * MASK_SIZE);
            bubbleSort(windowG, MASK_SIZE * MASK_SIZE);
            bubbleSort(windowB, MASK_SIZE * MASK_SIZE);

            // Assign the median value to the output pixel
            outputImage[index] = windowR[MASK_SIZE * MASK_SIZE / 2];
            outputImage[index + 1] = windowG[MASK_SIZE * MASK_SIZE / 2];
            outputImage[index + 2] = windowB[MASK_SIZE * MASK_SIZE / 2];
        }
        else
        {
            // If the pixel is on the image boundary, just copy the input to the output
            outputImage[index] = inputImage[index];
        }
    }
}

int main()
{
    int width, height, channels;
    unsigned char *inputImage = stbi_load("inp4.jpg", &width, &height, &channels, 3);
    size_t imageSize = width * height * channels * sizeof(unsigned char);

    // Allocate device memory for input and output images
    unsigned char *d_inputImage, *d_outputImage;
    hipMalloc((void **)&d_inputImage, imageSize);
    hipMalloc((void **)&d_outputImage, imageSize);

    hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

    // record time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Define grid and block dimensions
    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Launch the median filter kernel
    medianFilterKernel<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, width, height, channels);
    hipDeviceSynchronize();
    // stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    // Copy the result back to host memory and do further processing or save to file
    unsigned char *outputImage = (unsigned char *)malloc(imageSize);
    hipMemcpy(outputImage, d_outputImage, imageSize, hipMemcpyDeviceToHost);
    stbi_write_jpg("output.jpg", width, height, 3, outputImage, 100);

    printf("Image processing complete.\n");
    printf("Processing time: %.3f milliseconds\n", elapsed);
    // Free device memory and clean up
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    free(outputImage);

    return 0;
}